#include "hip/hip_runtime.h"
// -*- C++ -*-
// matrixMultiplication.cc
// a huge comparison of doing naive and tiled matrix multiplication using many
//  different methods and technologies

#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <string>
#include <algorithm>

// yucky, but for asking the system how many cores we have
#include <unistd.h>

// header file for openmp
#include <omp.h>

// header files for kokkos
#include <Kokkos_Core.hpp>
#include "Teuchos_Array.hpp"
#include "Intrepid_ArrayTools.hpp"
#include "Intrepid_FieldContainer.hpp"
#include "Intrepid_RealSpaceTools.hpp"
#include "Teuchos_oblackholestream.hpp"
#include "Teuchos_RCP.hpp"
#include "Teuchos_ScalarTraits.hpp"
#include "Teuchos_GlobalMPISession.hpp"

#include <hip/hip_runtime.h>

using std::string;
using std::vector;
using Intrepid::FieldContainer;

typedef Intrepid::RealSpaceTools<double> rst;

#define BLOCK_SIZE 64;

//Pre-C++11 timing (thanks jeff)
double getElapsedTime(const timespec start, const timespec end) {
	timespec temp;
	if ((end.tv_nsec-start.tv_nsec)<0) {
		temp.tv_sec = end.tv_sec-start.tv_sec-1;
		temp.tv_nsec = 1000000000+end.tv_nsec-start.tv_nsec;
	} else {
		temp.tv_sec = end.tv_sec-start.tv_sec;
		temp.tv_nsec = end.tv_nsec-start.tv_nsec;
	}
	return double(temp.tv_sec) + double(temp.tv_nsec) / 1e9;
}

__global__
void
cudaDoContractFieldFieldVector_kernel(const double * const __restrict__ d_left, const double * const __restrict__ d_right,
double * d_out,
int numCells,
int numPoints,
int dimVec,
int numLeftFields,
int numRightFields) {

	int myID = (blockIdx.x * blockDim.x) + threadIdx.x;

	if(myID < (numCells * numLeftFields * numRightFields)) {
		int matrixIndex = elementIndex % numCells;
		int rbf = matrixIndex % numRightFields;
		int lbf = matrixIndex % numLeftFields;

		double tmpVal = 0;
        for (int qp = 0; qp < numPoints; qp++) {
            for (int iVec = 0; iVec < dimVec; iVec++) {
                tmpVal += leftFields(cl, qp, iVec, lbf)*rightFields(cl, qp, iVec, rbf);
            } //D-loop
        } // P-loop
        outputFields(cl, lbf, rbf) = tmpVal;
	}
}

void
cudaDoContractFieldFieldVector(double * h_out,
		double * h_inLeft,
		double * h_inRight,
		int numCells,
		int numPoints,
		int dimVec,
		int numLeftFields,
		int numRightFields,
		timespec * tic,
		timespec * toc) {

	double * d_right;
	double * d_left;
	double * d_out;

	hipMalloc(&d_right, sizeof(double) * numCells  * numPoints * numRightFields * dimVec);

	hipMalloc(&d_left, sizeof(double) * numCells * numPoints * numLeftFields * dimVec);

	hipMalloc(&d_out, sizeof(double) * numCells * numRightFields * numLeftFields);

	hipMemset(d_out, 0, sizeof(double) * numCells * numRightFields * numLeftFields);

	hipMemcpy(d_right, h_inRight,
			sizeof(double) * numCells * numPoints * numRightFields, hipMemcpyHostToDevice);

	hipMemcpy(d_left, h_inLeft,
			sizeof(double) * numCells * numPoints * numLeftFields, hipMemcpyHostToDevice);


	dim3 blockSize(1024);
	dim3 gridSize((numCells * numLeftFields * numRightFields / 1024) + 1);
	
	clock_gettime(CLOCK_MONOTONIC, tic);
	cudaDoContractFieldFieldVector_kernel<<<gridSize, blockSize>>>(d_left,
			d_right, d_out, numCells, numPoints, dimVec, numLeftFields, numRightFields);
	
	hipDeviceSynchronize();
	clock_gettime(CLOCK_MONOTONIC, toc);
	hipMemcpy(h_out, d_out, sizeof(double) * numCells * numLeftFields * numRightFields, hipMemcpyDeviceToHost);

}

template<class DeviceType, class LeftViewType, class RightViewType, class OutputViewType>
struct ContractFieldFieldVectorFunctor {
	typedef DeviceType device_type;
	LeftViewType _leftFields;
	RightViewType _rightFields;
	OutputViewType _outputFields;
	int _numCells;
	int _numPoints;
	int _numLeftFields;
	int _numRightFields;
	int _dimVec;

	ContractFieldFieldVectorFunctor(LeftViewType leftFields,
			RightViewType rightFields,
			OutputViewType outputFields,
			int numCells,
			int numPoints,
			int dimVec,
			int numLeftFields,
			int numRightFields) :
		_leftFields(leftFields),
		_rightFields(rightFields),
		_outputFields(outputFields),
		_numPoints(numPoints),
		_numLeftFields(numLeftFields),
		_numRightFields(numRightFields),
		_dimVec(dimVec)
	{
		// Nothing to do
	}

	KOKKOS_INLINE_FUNCTION
		void operator()(const unsigned int elementIndex) const {
			
			int matrixIndex = elementIndex % _numCells;
			int rbf = matrixIndex % _numRightFields;
			int lbf = matrixIndex % _numLeftFields;

			double tmpVal = 0;
            for (int qp = 0; qp < numPoints; qp++) {
                for (int iVec = 0; iVec < dimVec; iVec++) {
                    tmpVal += leftFields(cl, qp, iVec, lbf)*rightFields(cl, qp, iVec, rbf);
                } //D-loop
            } // P-loop
            outputFields(cl, lbf, rbf) = tmpVal;
		}
};

// Serial ContractFieldFieldVector.  Contracts FieldContainers of doubles.
void ContractFieldFieldVectorSerial(FieldContainer<double> &  outputFields,
		const FieldContainer<double> &              leftFields,
		const FieldContainer<double> &              rightFields,
		double *                                    time = 0) {

	int numCells       = inputFields.dimension(0);
    int numFields      = inputFields.dimension(1);
    int numPoints      = inputFields.dimension(2);
    int dim1Tens       = inputFields.dimension(3);
    int dim2Tens       = inputFields.dimension(4);
    int numDataPoints  = inputData.dimension(1);

	if (numDataPoints != 1) { // nonconstant data
        for (int cl = 0; cl < numCells; cl++) {
            for (int lbf = 0; lbf < numFields; lbf++) {
                double tmpVal = 0;
                for (int qp = 0; qp < numPoints; qp++) {
                    for (int iTens1 = 0; iTens1 < dim1Tens; iTens1++) {
                        for (int iTens2 =0; iTens2 < dim2Tens; iTens2++) {
                          tmpVal += inputFields(cl, lbf, qp, iTens1, iTens2)*inputData(cl, qp, iTens1, iTens2);
                        } // D2-loop
                    } // D1-loop
                } // P-loop
                outputFields(cl, lbf) = tmpVal;
            } // F-loop
        } // C-loop
    }
    else { // constant data
        for (int cl = 0; cl < numCells; cl++) {
            for (int lbf = 0; lbf < numFields; lbf++) {
                Scalar tmpVal(0);
                for (int qp = 0; qp < numPoints; qp++) {
                    for (int iTens1 = 0; iTens1 < dim1Tens; iTens1++) {
                        for (int iTens2 = 0; iTens2 < dim2Tens; iTens2++) {
                            tmpVal += inputFields(cl, lbf, qp, iTens1, iTens2)*inputData(cl, 0, iTens1, iTens2);
                        } // D2-loop
                    } // D1-loop
                } // P-loop
                outputFields(cl, lbf) = tmpVal;
            } // F-loop
        } // C-loop
    } // numDataPoints
}


/*
 * Kokkos Cuda ContractFieldFieldVector.
 *
 * Contracts two Kokkos Cuda host views (two double *** tensors -> one double
 * *** tensor). Since
 *
 * Note that all input and output is in Kokkos host views --- the user is
 * responsible for getting the data in and out of them.
 */
template <class DeviceType, class input_view_t, class output_view_t, class input_host_t, class output_host_t>
void ContractFieldFieldVectorKokkos(output_host_t &   outHost,
		const input_host_t &                      leftHost,
		const input_host_t &                      rightHost,
		output_view_t &                           outDevice,
		input_view_t &                            leftDevice,
		input_view_t &                            rightDevice,
		double *                                  time = 0) {

	// get sizes
	int numCells        = leftFields.dimension(0);
    int numPoints       = leftFields.dimension(1);
    int dimVec          = leftFields.dimension(2);
    int numLeftFields   = leftFields.dimension(3);
    int numRightFields  = rightFields.dimension(3);

	// Deep copy Kokkos host views into device views
	Kokkos::deep_copy(leftDevice, leftHost);
	Kokkos::deep_copy(rightDevice, rightHost);
	Kokkos::deep_copy(outDevice, outHost);

	timespec tic;
	if(time != 0)
		clock_gettime(CLOCK_MONOTONIC, &tic);

	ContractFieldFieldVectorFunctor<DeviceType, input_view_t, input_view_t, output_view_t>
		kokkosFunctor(leftDevice, rightDevice, outDevice, numCells, numPoints, dimVec,
		numLeftFields, numRightFields);

	Kokkos::parallel_for(numCells * numRightFields * numLeftFields, kokkosFunctor);

	Kokkos::fence();

	timespec toc;
	if(time !=0){
		clock_gettime(CLOCK_MONOTONIC, &toc);
		*time += getElapsedTime(tic, toc);
	}

	Kokkos::deep_copy(outHost, outDevice);
}



int main(int argc, char* argv[]) {
	int c=10000, l=10, p=10, d1=10, d2 = 10, q = 10;

	FieldContainer<double> in_c_l_q_d1_d2(c, l, q, d1,d2);
	FieldContainer<double> in_c_q_d1_d2(c, q, d1, d2);
	FieldContainer<double> out1_c_l(c, l);
	FieldContainer<double> out2_c_l(c, l);
	double zero = Intrepid::INTREPID_TOL*100000.0;

	// fill with random numbers
	for (int i=0; i<in_c_l_q_d1_d2.size(); i++) {
		in_c_l_p[i] = Teuchos::ScalarTraits<double>::random();
	}
	for (int i=0; i<in_c_q_d1_d2.size(); i++) {
		in_c_r_p[i] = Teuchos::ScalarTraits<double>::random();
	}
	std::cout << "Created vectors" << std::endl;

	// ===============================================================
	// ********************** < Kokkos setup> ************************
	// vvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvv

	// Doing all of this here might throw off the timing -- we're not counting the
	// cost of the copy into Kokkos or the deep copy from Kokkos host to Kokkos
	// device.

	Kokkos::initialize();

	// Kokkos Cuda views
	typedef Kokkos::View<double *****, Kokkos::Cuda> cuda_input_view_t;
	typedef Kokkos::View<double ****, Kokkos::Cuda> cuda_output_view_t;
	typedef typename cuda_input_view_t::HostMirror cuda_input_host_t;
	typedef typename cuda_output_view_t::HostMirror cuda_output_host_t;

	// Kokkos OpenMP views
	typedef Kokkos::View<double *****, Kokkos::OpenMP> omp_input_view_t;
	typedef Kokkos::View<double ****, Kokkos::OpenMP> omp_output_view_t;
	typedef typename omp_input_view_t::HostMirror omp_input_host_t;
	typedef typename omp_output_view_t::HostMirror omp_output_host_t;


	//Cuda arrays

	double * cudaRight = new double[c * l * q * d1 * d2];
	double * cudaLeft = new double[c * q * d1 * d2];

	double * cudaOut = new double[c * l];


	// Make equivalent Kokkos views 
	cuda_input_view_t cuda_kokkosLeft("left_input", c,q,d1,d2,l);
	cuda_input_view_t cuda_kokkosRight("right_input",q,d1,d2, c);
	cuda_output_view_t cuda_kokkosOut("output", c, l);

	omp_input_view_t omp_kokkosLeft("left_input", c,q,d1,d2,l);
	omp_input_view_t omp_kokkosRight("right_input", q,d1,d2, c);
	omp_output_view_t omp_kokkosOut("output", c,l);

	// And their host mirrors

	cuda_input_host_t cuda_hostLeft = Kokkos::create_mirror_view(cuda_kokkosLeft);
	cuda_input_host_t cuda_hostRight = Kokkos::create_mirror_view(cuda_kokkosRight);
	cuda_output_host_t cuda_hostOut = Kokkos::create_mirror_view(cuda_kokkosOut);

	omp_input_host_t omp_hostLeft = Kokkos::create_mirror_view(omp_kokkosLeft);
	omp_input_host_t omp_hostRight = Kokkos::create_mirror_view(omp_kokkosRight);
	omp_output_host_t omp_hostOut = Kokkos::create_mirror_view(omp_kokkosOut);

	// Copy into Kokkos host views and cuda
	for (int cl = 0; cl < c; ++cl) {
		for (int qp = 0; qp < p; ++qp) {
			for(int dim1 = 0; dim1 < d1; ++dim1){
				for(int dim2 = 0; dim2 < d2; ++dim2) {
					cuda_hostRight(qp, dim1, dim2, cl) = in_c_q_d1_d2(cl,qp,dim1,dim2);
					omp_hostRight(qp, dim1, dim2, cl) = in_c_q_d1_d2(cl,qp,dim1,dim2);

					cudaRight[qp * c * d1 * d2 + dim1 * d2 * c + dim2 * c + cl] = in_c_q_d1_d2(cl,qp,dim1,dim2);
				}
			
				for(int lbf = 0; lbf < l; ++lbf) {
					cuda_hostLeft(cl,qp, dim1, dim2, lbf) = in_c_l_q_d1_d2(cl,lbf,qp,dim1,dim2);
					omp_hostLeft(cl,qp, dim1, dim2, lbf) = in_c_l_q_d1_d2(cl,lbf,qp,dim1,dim2);

					cudaLeft[cl * p * d1 * d2 * l + qp * d1 * d2 * l + dim1 * d2 * l + dim2 * l + lbf] = in_c_l_q_d1_d2(cl,lbf,qp,dim1,dim2);
				}
			}
		}
	}



	// ===============================================================
	// ********************** </Kokkos setup> ************************
	// vvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvv

	std::cout << "trying serial" << std::endl;

	//Warmup
	ContractFieldFieldVectorSerial(out2_c_l, in_c_l_q_d1_d2, in_c_r_p);

	timespec tic;
	clock_gettime(CLOCK_MONOTONIC, &tic);

	//repeat the calculation 5 times so we can average out some randomness
	for(int i = 0; i < 5; ++i){
		ContractFieldFieldVectorSerial(out2_c_l, in_c_l_q_d1_d2, in_c_r_p);
	}

	timespec toc;
	clock_gettime(CLOCK_MONOTONIC, &toc);
	const double elapsedTime_serial = getElapsedTime(tic, toc);

	printf("trying kokkos openmp\n");

	//Warmpup
	ContractFieldFieldVectorKokkos<Kokkos::OpenMP, omp_input_view_t,
		omp_output_view_t, omp_input_host_t, omp_output_host_t>
			(omp_hostOut, omp_hostLeft, omp_hostRight, omp_kokkosOut,
			 omp_kokkosLeft, omp_kokkosRight);
	clock_gettime(CLOCK_MONOTONIC, &tic);
	//repeat the calculation 5 times so we can average out some randomness
	for(int i = 0; i < 5; ++i){
		ContractFieldFieldVectorKokkos<Kokkos::OpenMP, omp_input_view_t,
			omp_output_view_t, omp_input_host_t, omp_output_host_t>
				(omp_hostOut, omp_hostLeft, omp_hostRight, omp_kokkosOut,
				 omp_kokkosLeft, omp_kokkosRight);
	}
	clock_gettime(CLOCK_MONOTONIC, &toc);
	const double elapsedTime_kokkos_omp = getElapsedTime(tic, toc);
	// Copy out from kokkos output view (NOT timing this)
	for (int cl = 0; cl < c; ++cl) {
		for(int lbf = 0; lbf < l; ++lbf) {
			out1_c_l(cl,lbf) = omp_hostOut(cl,lbf);
		}
	}
	rst::subtract(&out1_c_l[0], &out2_c_l[0], out2_c_l.size());
	if (rst::vectorNorm(&out1_c_l[0], out1_c_l.size(), Intrepid::NORM_ONE) > zero) {
		std::cout << "\n\nINCORRECT contractFieldFieldTensor (1): check COMP_CPP vs. COMP_KOKKOS; "
			<< " diff-1norm = " << rst::vectorNorm(&out1_c_l[0], out1_c_l.size(), Intrepid::NORM_ONE) << "\n\n";
	}
	std::cout << "kokkos omp speedup of " << elapsedTime_serial/elapsedTime_kokkos_omp << std::endl;


	printf("trying kokkos cuda\n");
 
	//Warmpup
	ContractFieldFieldVectorKokkos<Kokkos::Cuda, cuda_input_view_t,
		cuda_output_view_t, cuda_input_host_t, cuda_output_host_t>
			(cuda_hostOut, cuda_hostLeft, cuda_hostRight, cuda_kokkosOut,
			 cuda_kokkosLeft, cuda_kokkosRight);
	clock_gettime(CLOCK_MONOTONIC, &tic);
	//repeat the calculation 5 times so we can average out some randomness
	for(int i = 0; i < 5; ++i){
		ContractFieldFieldVectorKokkos<Kokkos::Cuda, cuda_input_view_t,
			cuda_output_view_t, cuda_input_host_t, cuda_output_host_t>
				(cuda_hostOut, cuda_hostLeft, cuda_hostRight, cuda_kokkosOut,
				 cuda_kokkosLeft, cuda_kokkosRight);
	}
	clock_gettime(CLOCK_MONOTONIC, &toc);
	const double elapsedTime_kokkos_cuda = getElapsedTime(tic, toc);
	// Copy out from kokkos output view (NOT timing this)
	for (int cl = 0; cl < c; ++cl) {
		for(int lbf = 0; lbf < l; ++lbf) {
			out1_c_l(cl,lbf) = omp_hostOut(cl,lbf);
		}
	}
	rst::subtract(&out1_c_l[0], &out2_c_l[0], out2_c_l.size());
	if (rst::vectorNorm(&out1_c_l[0], out1_c_l.size(), Intrepid::NORM_ONE) > zero) {
		std::cout << "\n\nINCORRECT contractFieldFieldTensor (0): check COMP_CPP vs. COMP_KOKKOS; "
			<< " diff-1norm = " << rst::vectorNorm(&out1_c_l[0], out1_c_l.size(), Intrepid::NORM_ONE) << "\n\n";
	}
	std::cout << "kokkos cuda speedup of " << elapsedTime_serial/elapsedTime_kokkos_cuda << std::endl;

	Kokkos::finalize();

	std::cout << "trying cuda major" << std::endl;
	//Now try the cuda version, start with warmup
	cudaDoContractFieldFieldVector(cudaOut,cudaLeft,cudaRight, c, p, d1, d2, l, &tic, &toc);
	double elapsedTime_cuda = 0;
	
	for(int i = 0; i < 5; ++i){
		cudaDoContractFieldFieldVector(cudaOut,cudaLeft,cudaRight, c, p, d1, d2, l, &tic, &toc);
		elapsedTime_cuda += getElapsedTime(tic,toc);
	}

	for (int cl = 0; cl < c; ++cl) {
		for(int lbf = 0; lbf < l; ++lbf) {
			out1_c_l(cl,lbf) = omp_hostOut(cl,lbf);
		}
	}

	rst::subtract(&out1_c_l[0], &out2_c_l[0], out2_c_l.size());
	if (rst::vectorNorm(&out1_c_l[0], out1_c_l.size(), Intrepid::NORM_ONE) > zero) {
		std::cout << "\n\nINCORRECT contractFieldFieldTensor (1): check cuda; "
		<< " diff-1norm = " << rst::vectorNorm(&out1_c_l[0], out1_c_l.size(), Intrepid::NORM_ONE) << "\n\n";
	}

	std::cout << "cuda speedup of " << elapsedTime_serial/elapsedTime_cuda << std::endl;


	return 0;
}