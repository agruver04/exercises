#include "hip/hip_runtime.h"
// -*- C++ -*-
// matrixMultiplication.cc
// a huge comparison of doing naive and tiled matrix multiplication using many
//  different methods and technologies

#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <string>
#include <algorithm>

// yucky, but for asking the system how many cores we have
#include <unistd.h>

// header file for openmp
#include <omp.h>

// header files for kokkos
#include <Kokkos_Core.hpp>
#include "Teuchos_Array.hpp"
#include "Intrepid_ArrayTools.hpp"
#include "Intrepid_FieldContainer.hpp"
#include "Intrepid_RealSpaceTools.hpp"
#include "Teuchos_oblackholestream.hpp"
#include "Teuchos_RCP.hpp"
#include "Teuchos_ScalarTraits.hpp"
#include "Teuchos_GlobalMPISession.hpp"

#include <hip/hip_runtime.h>

using std::string;
using std::vector;
using Intrepid::FieldContainer;

typedef Intrepid::RealSpaceTools<double> rst;

#define BLOCK_SIZE 64;

//Pre-C++11 timing (thanks jeff)
double getElapsedTime(const timespec start, const timespec end) {
	timespec temp;
	if ((end.tv_nsec-start.tv_nsec)<0) {
		temp.tv_sec = end.tv_sec-start.tv_sec-1;
		temp.tv_nsec = 1000000000+end.tv_nsec-start.tv_nsec;
	} else {
		temp.tv_sec = end.tv_sec-start.tv_sec;
		temp.tv_nsec = end.tv_nsec-start.tv_nsec;
	}
	return double(temp.tv_sec) + double(temp.tv_nsec) / 1e9;
}


__global__
void
cudaDocontractFieldFieldScalar_kernelColMajor(double * d_left, double * d_right,
		double * d_out,
		int numCells,
		int numPoints) {

	int myID = (blockIdx.x * blockDim.x) + threadIdx.x;

	if(myID < numCells) {
		double temp = 0;
		for (int qp = 0; qp < numPoints; qp++) {
			temp += d_left[myID + qp*numCells] * d_right[myID + qp*numCells];
		}
		d_out[myID]=temp;
	}
}

__global__
void
cudaDocontractFieldFieldScalar_kernel(const double * const __restrict__ d_left, const double * const __restrict__ d_right,
double * d_out,
int numCells,
int numLeftFields,
int numRightFields,
int numPoints) {

	int myID = (blockIdx.x * blockDim.x) + threadIdx.x;

	if(myID < (numCells * numLeftFields * numRightFields)) {
		int myMatrix = myID / (numLeftFields * numRightFields);
		int matrixIndex = myID % (numLeftFields * numRightFields);

		int matrixRow = matrixIndex / numRightFields;
		int matrixCol = matrixIndex % numRightFields;

		double temp = 0;
		for (int qp = 0; qp < numPoints; qp++) {
			temp += d_left[myMatrix*numPoints*numLeftFields + numPoints*matrixRow + qp] *
							d_right[myMatrix*numPoints*numRightFields + qp*numRightFields + matrixCol];
		}
		d_out[myID]= temp;
	}
}

void
cudaDocontractFieldFieldScalar(double * h_out,
		double * h_inLeft,
		double * h_inRight,
		int numCells,
		int numLeftFields,
		int numRightFields,
		int numPoints,
		timespec * tic,
		timespec * toc) {

	double * d_right;
	double * d_left;
	double * d_out;

	hipMalloc(&d_right, sizeof(double) * numCells  * numPoints * numRightFields);

	hipMalloc(&d_left, sizeof(double) * numCells * numPoints * numLeftFields);

	hipMalloc(&d_out, sizeof(double) * numCells * numRightFields * numLeftFields);

	hipMemset(d_out, 0, sizeof(double) * numCells * numRightFields * numLeftFields);

	hipMemcpy(d_right, h_inRight,
			sizeof(double) * numCells * numPoints * numRightFields, hipMemcpyHostToDevice);

	hipMemcpy(d_left, h_inLeft,
			sizeof(double) * numCells * numPoints * numLeftFields, hipMemcpyHostToDevice);


	dim3 blockSize(1024);
	dim3 gridSize((numCells * numLeftFields * numRightFields / 1024) + 1);
	
	clock_gettime(CLOCK_MONOTONIC, tic);
	cudaDocontractFieldFieldScalar_kernel<<<gridSize, blockSize>>>(d_left,
			d_right, d_out, numCells, numLeftFields, numRightFields, numPoints);
	
	hipDeviceSynchronize();
	clock_gettime(CLOCK_MONOTONIC, toc);
	hipMemcpy(h_out, d_out, sizeof(double) * numCells * numLeftFields * numRightFields, hipMemcpyDeviceToHost);

}
/*
__global__
void
cudaDocontractFieldFieldScalar_kernelUnrolled(double * d_left, double * d_right,
double * d_out,
int numCells,
int numPoints) {
	int myID = (blockIdx.x * blockDim.x) + threadIdx.x + (blockIdx.y * numPoints);
	if(myID < numCells * numPoints) {
		temp += d_left[myID*numPoints + qp] * d_right[myID*numPoints + qp];
		d_out[myID]= temp;
	}
}
*/
/*
void
cudaDocontractFieldFieldScalarUnrolled(double * h_out,
double * h_inLeft,
double * h_inRight,
int numCells,
int numPoints,
bool colMajor) {
	double * d_right;
	double * d_left;
	double * d_out;
	hipMalloc(&d_right, sizeof(double) * numCells  * numPoints);
	hipMalloc(&d_left, sizeof(double) * numCells * numPoints);
	hipMalloc(&d_out, sizeof(double) * numCells);
	hipMemset(d_out, 0, sizeof(double) * numCells);
	hipMemcpy(d_right, h_inRight,
	sizeof(double) * numCells * numPoints, hipMemcpyHostToDevice);
	hipMemcpy(d_left, h_inLeft,
	sizeof(double) * numCells * numPoints, hipMemcpyHostToDevice);
	dim3 blockSize(64);
	dim3 gridSize((numPoints / 64) + 1, (numCells / 64) + 1);
	cudaDocontractFieldFieldScalar_kernelUnrolled<<<gridSize, blockSize>>>(d_left,
	d_right, d_out, numCells,numPoints);
	hipMemcpy(h_out, d_out, sizeof(double) * numCells, hipMemcpyDeviceToHost);
}
*/
template<class DeviceType, class LeftViewType, class RightViewType, class OutputViewType>
struct contractFieldFieldScalarFunctor {
	typedef DeviceType device_type;
	LeftViewType _leftFields;
	RightViewType _rightFields;
	OutputViewType _outputFields;
	int _numPoints;
	int _numLeftFields;
	int _numRightFields;

	contractFieldFieldScalarFunctor(LeftViewType leftFields,
			RightViewType rightFields,
			OutputViewType outputFields,
			int numLeftFields,
			int numRightFields,
			int numPoints) :
		_leftFields(leftFields),
		_rightFields(rightFields),
		_outputFields(outputFields),
		_numPoints(numPoints),
		_numLeftFields(numLeftFields),
		_numRightFields(numRightFields)
	{
		// Nothing to do
	}

	KOKKOS_INLINE_FUNCTION
		void operator()(const unsigned int elementIndex) const {
			for (int lbf = 0; lbf < _numLeftFields; lbf++) {
				for (int rbf = 0; rbf < _numRightFields; rbf++) {
					double tmpVal = 0;
					for (int qp = 0; qp < _numPoints; qp++) {
						tmpVal += _leftFields(elementIndex, lbf, qp)*_rightFields(elementIndex, rbf, qp);
					} // P-loop
					_outputFields(elementIndex, lbf, rbf) = tmpVal;
				} // R-loop
			} // L-loop
		}
};




template<class DeviceType, class LeftViewType, class RightViewType, class OutputViewType>
struct contractFieldFieldScalarFunctor1D {
	typedef DeviceType device_type;
	LeftViewType _leftFields;
	RightViewType _rightFields;
	OutputViewType _outputFields;
	int _numLeftFields;
	int _numRightFields;
	int _numPoints;
	int _dim1Tensor;
	int _dim2Tensor;
	int _numCells;

	contractFieldFieldScalarFunctor1D(LeftViewType leftFields,
			RightViewType rightFields,
			OutputViewType outputFields,
			int numLeftFields,
			int numRightFields,
			int numPoints,
			int dim1Tensor,
			int dim2Tensor,
			int numCells) :
		_leftFields(leftFields),
		_rightFields(rightFields),
		_outputFields(outputFields),
		_numLeftFields(numLeftFields),
		_numRightFields(numRightFields),
		_numPoints(numPoints),
		_dim1Tensor(dim1Tensor),
		_dim2Tensor(dim2Tensor),
		_numCells(numCells)
	{
		// Nothing to do
	}

	KOKKOS_INLINE_FUNCTION
		void operator()(const unsigned int elementIndex) const {

			for (int lbf = 0; lbf < _numLeftFields; lbf++) {
				for (int rbf = 0; rbf < _numRightFields; rbf++) {
					double tmpVal = 0;
					for (int qp = 0; qp < _numPoints; qp++) {
						for (int iTens1 = 0; iTens1 < _dim1Tensor; iTens1++) {
							for (int iTens2 = 0; iTens2 < _dim2Tensor; iTens2++) {
								tmpVal +=
									_leftFields(lbf*_numPoints*_dim1Tensor*_dim2Tensor*_numCells +
											qp*_dim1Tensor*_dim2Tensor*_numCells +
											iTens1*_dim2Tensor*_numCells + iTens2*_numCells + elementIndex)
									*_rightFields(rbf*_numPoints*_dim1Tensor*_dim2Tensor*_numCells +
											qp*_dim1Tensor*_dim2Tensor*_numCells +
											iTens1*_dim2Tensor*_numCells + iTens2*_numCells + elementIndex);
							} // D2-loop
						} // D1-loop
					} // P-loop
					_outputFields(lbf*_numRightFields*_numCells +
							rbf*_numCells + elementIndex) = tmpVal;
				} // R-loop
			} // L-loop
		}
};





// Serial contractFieldFieldScalar.  Contracts FieldContainers of doubles.
void contractFieldFieldScalarSerial(FieldContainer<double> &  outputFields,
		const FieldContainer<double> &              leftFields,
		const FieldContainer<double> &              rightFields,
		double *                                    time = 0) {

	// TODO(ellen): Might later want to template this so that both the container
	//              and the scalars inside the container are template arguments,
	//              so we can hand it kokkos views or custom structs.
	int numCells        = leftFields.dimension(0);
	int numLeftFields   = leftFields.dimension(1);
	int numRightFields  = rightFields.dimension(1);
	int numPoints       = leftFields.dimension(2);

	for (int cl = 0; cl < numCells; cl++) {
		for (int lbf = 0; lbf < numLeftFields; lbf++) {
			for (int rbf = 0; rbf < numRightFields; rbf++) {
				double tmpVal = 0;
				for (int qp = 0; qp < numPoints; qp++) {
					tmpVal += leftFields(cl, lbf, qp)*rightFields(cl, rbf, qp);
				} // P-loop
				outputFields(cl, lbf, rbf) = tmpVal;
			} // R-loop
		} // L-loop
	} // C-loop
}


/*
 * Kokkos Cuda contractFieldFieldScalar.
 *
 * Contracts two Kokkos Cuda host views (two double *** tensors -> one double
 * *** tensor). Since
 *
 * Note that all input and output is in Kokkos host views --- the user is
 * responsible for getting the data in and out of them.
 */
template <class DeviceType, class input_view_t, class output_view_t, class input_host_t, class output_host_t>
void contractFieldFieldScalarKokkos(output_host_t &   outHost,
		const input_host_t &                      leftHost,
		const input_host_t &                      rightHost,
		output_view_t &                           outDevice,
		input_view_t &                            leftDevice,
		input_view_t &                            rightDevice,
		double *                                  time = 0) {

	// get sizes
	int numCells        = leftHost.dimension(0);
	int numLeftFields   = leftHost.dimension(1);
	int numRightFields  = rightHost.dimension(1);
	int numPoints       = leftHost.dimension(2);

	// Deep copy Kokkos host views into device views
	Kokkos::deep_copy(leftDevice, leftHost);
	Kokkos::deep_copy(rightDevice, rightHost);
	Kokkos::deep_copy(outDevice, outHost);

	timespec tic;
	if(time != 0)
		clock_gettime(CLOCK_MONOTONIC, &tic);

	contractFieldFieldScalarFunctor<DeviceType, input_view_t, input_view_t, output_view_t>
		kokkosFunctor(leftDevice, rightDevice, outDevice, numLeftFields,
		numRightFields, numPoints);

	Kokkos::parallel_for(numCells, kokkosFunctor);

	Kokkos::fence();

	timespec toc;
	if(time !=0){
		clock_gettime(CLOCK_MONOTONIC, &toc);
		*time += getElapsedTime(tic, toc);
	}

	Kokkos::deep_copy(outHost, outDevice);
}


template <class DeviceType, class input_view_t, class output_view_t, class input_host_t, class output_host_t>
void contractFieldFieldScalarKokkos1D(output_host_t &   outHost,
		const input_host_t &                      leftHost,
		const input_host_t &                      rightHost,
		output_view_t &                           outDevice,
		input_view_t &                            leftDevice,
		input_view_t &                            rightDevice,
		int   numCells,
		int numLeftFields,
		int numRightFields,
		int numPoints,
		int dim1Tensor,
		int dim2Tensor,
		double *                                  time = 0
		) {
	/*
	// get sizes
	int numCells        = leftHost.dimension(0);
	int numLeftFields   = leftHost.dimension(1);
	int numRightFields  = rightHost.dimension(1);
	int numPoints       = leftHost.dimension(2);
	int dim1Tensor      = leftHost.dimension(3);
	int dim2Tensor      = leftHost.dimension(4);
	 */


	// Deep copy Kokkos host views into device views
	Kokkos::deep_copy(leftDevice, leftHost);
	Kokkos::deep_copy(rightDevice, rightHost);
	Kokkos::deep_copy(outDevice, outHost);

	timespec tic;
	if(time != 0)
		clock_gettime(CLOCK_MONOTONIC, &tic);

	contractFieldFieldScalarFunctor1D<DeviceType, input_view_t, input_view_t, output_view_t>
		kokkosFunctor(leftDevice, rightDevice, outDevice, numLeftFields,
				numRightFields, numPoints, dim1Tensor, dim2Tensor, numCells);

	Kokkos::parallel_for(numCells, kokkosFunctor);

	Kokkos::fence();

	timespec toc;
	if(time !=0){
		clock_gettime(CLOCK_MONOTONIC, &toc);
		*time += getElapsedTime(tic, toc);
	}

	Kokkos::deep_copy(outHost, outDevice);
}



int main(int argc, char* argv[]) {
	int c=10000, p=10, l=10, r=10;

	FieldContainer<double> in_c_l_p(c, l, p);
	FieldContainer<double> in_c_r_p(c, r, p);
	FieldContainer<double> out1_c_l_r(c, l, r);
	FieldContainer<double> out2_c_l_r(c, l, r);
	double zero = Intrepid::INTREPID_TOL*100000.0;

	// fill with random numbers
	for (int i=0; i<in_c_l_p.size(); i++) {
		in_c_l_p[i] = Teuchos::ScalarTraits<double>::random();
	}
	for (int i=0; i<in_c_r_p.size(); i++) {
		in_c_r_p[i] = Teuchos::ScalarTraits<double>::random();
	}
	std::cout << "Created vectors" << std::endl;

	// ===============================================================
	// ********************** < Kokkos setup> ************************
	// vvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvv

	// Doing all of this here might throw off the timing -- we're not counting the
	// cost of the copy into Kokkos or the deep copy from Kokkos host to Kokkos
	// device.

	Kokkos::initialize();

	// Kokkos Cuda views
	typedef Kokkos::View<double ***, Kokkos::Cuda> cuda_input_view_t;
	typedef Kokkos::View<double ***, Kokkos::Cuda> cuda_output_view_t;
	typedef typename cuda_input_view_t::HostMirror cuda_input_host_t;
	typedef typename cuda_output_view_t::HostMirror cuda_output_host_t;

	// Kokkos OpenMP views
	typedef Kokkos::View<double ***, Kokkos::OpenMP> omp_input_view_t;
	typedef Kokkos::View<double ***, Kokkos::OpenMP> omp_output_view_t;
	typedef typename omp_input_view_t::HostMirror omp_input_host_t;
	typedef typename omp_output_view_t::HostMirror omp_output_host_t;


	//Cuda arrays

	double * cudaRight = new double[c * r * p];
	double * cudaLeft = new double[c * l * p];

	double * cudaOut = new double[c * l * r];


	// Make equivalent Kokkos views

	cuda_input_view_t cuda_kokkosLeft("left_input", c,l, p);
	cuda_input_view_t cuda_kokkosRight("right_input", c, r, p);
	cuda_output_view_t cuda_kokkosOut("output", c, l, r );

	omp_input_view_t omp_kokkosLeft("left_input", c, l, p);
	omp_input_view_t omp_kokkosRight("right_input",  c,r, p);
	omp_output_view_t omp_kokkosOut("output", c,l ,r);

	// And their host mirrors

	cuda_input_host_t cuda_hostLeft = Kokkos::create_mirror_view(cuda_kokkosLeft);
	cuda_input_host_t cuda_hostRight = Kokkos::create_mirror_view(cuda_kokkosRight);
	cuda_output_host_t cuda_hostOut = Kokkos::create_mirror_view(cuda_kokkosOut);

	omp_input_host_t omp_hostLeft = Kokkos::create_mirror_view(omp_kokkosLeft);
	omp_input_host_t omp_hostRight = Kokkos::create_mirror_view(omp_kokkosRight);
	omp_output_host_t omp_hostOut = Kokkos::create_mirror_view(omp_kokkosOut);

	// Copy into Kokkos host views and cuda
	// Need to change this so that its 1-D and cl has stride 1
	for (int cl = 0; cl < c; ++cl) {
		for (int qp = 0; qp < p; ++qp) {
			for(int rbf = 0; rbf < r; ++rbf) {
				cuda_hostRight(cl,rbf, qp) = in_c_r_p(cl,rbf,qp);
				omp_hostRight(cl,rbf,qp) = in_c_r_p(cl,rbf,qp);

				cudaRight[cl * p * r + r * qp + rbf] = in_c_r_p(cl,rbf,qp);
			}
			for(int lbf = 0; lbf < l; ++lbf) {
				cuda_hostLeft(cl, lbf, qp) = in_c_l_p(cl,lbf, qp);
				omp_hostLeft(cl,lbf, qp) = in_c_l_p(cl,lbf,qp);

				cudaLeft[cl * p * l + p * lbf + qp] = in_c_l_p(cl,lbf,qp);
			}
			//cudaRightColMajor[cl + c*qp] = in_r_c_p(cl,qp);
			//cudaLeftColMajor[cl + c*qp] = in_l_c_p(cl,qp);

			//cudaRightRowMajor[cl * p + qp] = in_r_c_p(cl,qp);
			//cudaLeftRowMajor[cl * p + qp] = in_l_c_p(cl,qp);
		}
	}



	// ===============================================================
	// ********************** </Kokkos setup> ************************
	// vvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvv

	std::cout << "trying serial" << std::endl;

	//Warmup
	contractFieldFieldScalarSerial(out2_c_l_r, in_c_l_p, in_c_r_p);

	timespec tic;
	clock_gettime(CLOCK_MONOTONIC, &tic);

	//repeat the calculation 5 times so we can average out some randomness
	for(int i = 0; i < 5; ++i){
		contractFieldFieldScalarSerial(out2_c_l_r, in_c_l_p, in_c_r_p);
	}

	timespec toc;
	clock_gettime(CLOCK_MONOTONIC, &toc);
	const double elapsedTime_serial = getElapsedTime(tic, toc);

	printf("trying kokkos openmp\n");
/*
	//Warmpup
	contractFieldFieldScalarKokkos<Kokkos::OpenMP, omp_input_view_t,
		omp_output_view_t, omp_input_host_t, omp_output_host_t>
			(omp_hostOut, omp_hostLeft, omp_hostRight, omp_kokkosOut,
			 omp_kokkosLeft, omp_kokkosRight);
	clock_gettime(CLOCK_MONOTONIC, &tic);
	//repeat the calculation 5 times so we can average out some randomness
	for(int i = 0; i < 5; ++i){
		contractFieldFieldScalarKokkos<Kokkos::OpenMP, omp_input_view_t,
			omp_output_view_t, omp_input_host_t, omp_output_host_t>
				(omp_hostOut, omp_hostLeft, omp_hostRight, omp_kokkosOut,
				 omp_kokkosLeft, omp_kokkosRight);
	}
	clock_gettime(CLOCK_MONOTONIC, &toc);
	const double elapsedTime_kokkos_omp = getElapsedTime(tic, toc);
	// Copy out from kokkos output view (NOT timing this)
	for (int cl = 0; cl < c; ++cl) {
		for(int lbf = 0; lbf < l; ++lbf) {
			for(int rbf = 0; rbf < r; ++rbf) {
				out1_c_l_r(cl,lbf,rbf) = omp_hostOut(cl,lbf,rbf);
			}
		}
	}
	rst::subtract(&out1_c_l_r[0], &out2_c_l_r[0], out2_c_l_r.size());
	if (rst::vectorNorm(&out1_c_l_r[0], out1_c_l_r.size(), Intrepid::NORM_ONE) > zero) {
		std::cout << "\n\nINCORRECT contractFieldFieldTensor (1): check COMP_CPP vs. COMP_KOKKOS; "
			<< " diff-1norm = " << rst::vectorNorm(&out1_c_l_r[0], out1_c_l_r.size(), Intrepid::NORM_ONE) << "\n\n";
	}
	std::cout << "kokkos omp speedup of " << elapsedTime_serial/elapsedTime_kokkos_omp << std::endl;
*/
/*
	printf("trying kokkos cuda\n");
 
	//Warmpup
	contractFieldFieldScalarKokkos<Kokkos::Cuda, cuda_input_view_t,
		cuda_output_view_t, cuda_input_host_t, cuda_output_host_t>
			(cuda_hostOut, cuda_hostLeft, cuda_hostRight, cuda_kokkosOut,
			 cuda_kokkosLeft, cuda_kokkosRight);
	clock_gettime(CLOCK_MONOTONIC, &tic);
	//repeat the calculation 5 times so we can average out some randomness
	for(int i = 0; i < 5; ++i){
		contractFieldFieldScalarKokkos<Kokkos::Cuda, cuda_input_view_t,
			cuda_output_view_t, cuda_input_host_t, cuda_output_host_t>
				(cuda_hostOut, cuda_hostLeft, cuda_hostRight, cuda_kokkosOut,
				 cuda_kokkosLeft, cuda_kokkosRight);
	}
	clock_gettime(CLOCK_MONOTONIC, &toc);
	const double elapsedTime_kokkos_cuda = getElapsedTime(tic, toc);
	// Copy out from kokkos output view (NOT timing this)
	for (int cl = 0; cl < c; ++cl) {
		for(int lbf = 0; lbf < l; ++lbf) {
			for(int rbf = 0; rbf < r; ++rbf) {
				out1_c_l_r(cl,lbf,rbf) = omp_hostOut(cl,lbf,rbf);
			}
		}
	}
	rst::subtract(&out1_c_l_r[0], &out2_c_l_r[0], out2_c_l_r.size());
	if (rst::vectorNorm(&out1_c_l_r[0], out1_c_l_r.size(), Intrepid::NORM_ONE) > zero) {
		std::cout << "\n\nINCORRECT contractFieldFieldTensor (0): check COMP_CPP vs. COMP_KOKKOS; "
			<< " diff-1norm = " << rst::vectorNorm(&out1_c_l_r[0], out1_c_l_r.size(), Intrepid::NORM_ONE) << "\n\n";
	}
	std::cout << "kokkos cuda speedup of " << elapsedTime_serial/elapsedTime_kokkos_cuda << std::endl;
*/
	Kokkos::finalize();

	std::cout << "trying cuda major" << std::endl;
	//Now try the cuda version, start with warmup
	cudaDocontractFieldFieldScalar(cudaOut,cudaLeft,cudaRight, c, l, r, p, &tic, &toc);
	double elapsedTime_cuda = 0;
	
	for(int i = 0; i < 5; ++i){
		cudaDocontractFieldFieldScalar(cudaOut,cudaLeft,cudaRight, c, l, r, p, &tic, &toc);
		elapsedTime_cuda += getElapsedTime(tic,toc);
	}

	for (int cl = 0; cl < c; ++cl) {
		for(int lbf = 0; lbf < l; ++lbf) {
			for(int rbf = 0; rbf < r; ++rbf) {
				out1_c_l_r(cl,lbf,rbf) = cudaOut[cl * l * r + lbf * r + rbf];
			}
		}
	}

	rst::subtract(&out1_c_l_r[0], &out2_c_l_r[0], out2_c_l_r.size());
	if (rst::vectorNorm(&out1_c_l_r[0], out1_c_l_r.size(), Intrepid::NORM_ONE) > zero) {
		std::cout << "\n\nINCORRECT contractFieldFieldTensor (1): check cuda; "
		<< " diff-1norm = " << rst::vectorNorm(&out1_c_l_r[0], out1_c_l_r.size(), Intrepid::NORM_ONE) << "\n\n";
	}

	std::cout << "cuda speedup of " << elapsedTime_serial/elapsedTime_cuda << std::endl;

	/*
	std::cout << "trying cuda row major" << std::endl;
	//Now try the cuda version, start with warmup
	cudaDocontractFieldFieldScalar(cudaOut,cudaLeftRowMajor,cudaRightRowMajor, c, p, 0);
	clock_gettime(CLOCK_MONOTONIC, &tic);
	for(int i = 0; i < 5; ++i){
		cudaDocontractFieldFieldScalar(cudaOut,cudaLeftRowMajor,cudaRightRowMajor, c, p, 0);
	}
	clock_gettime(CLOCK_MONOTONIC, &toc);
	const double elapsedTime_cudaRow = getElapsedTime(tic, toc);
	for (int cl = 0; cl < c; ++cl) {
			out1_c_l_r(cl) = cudaOut[cl];
	}
	rst::subtract(&out1_c_l_r[0], &out2_c_l_r[0], out2_c_l_r.size());
	if (rst::vectorNorm(&out1_c_l_r[0], out1_c_l_r.size(), Intrepid::NORM_ONE) > zero) {
		std::cout << "\n\nINCORRECT contractFieldFieldTensor (1): check cuda; "
		<< " diff-1norm = " << rst::vectorNorm(&out1_c_l_r[0], out1_c_l_r.size(), Intrepid::NORM_ONE) << "\n\n";
	}
	std::cout << "cuda row major speedup of " << elapsedTime_serial/elapsedTime_cudaRow << std::endl;
	std::cout << "trying cuda thread-per-index" << std::endl;
	//Now try the cuda version, start with warmup
	cudaDocontractFieldFieldScalarUnrolled(cudaOut,cudaLeftRowMajor,cudaRightRowMajor, c, p);
	clock_gettime(CLOCK_MONOTONIC, &tic);
	for(int i = 0; i < 5; ++i){
		cudaDocontractFieldFieldScalarUnrolled(cudaOut,cudaLeftRowMajor,cudaRightRowMajor, c, p);
	}
	clock_gettime(CLOCK_MONOTONIC, &toc);
	const double elapsedTime_cudaUnrolled = getElapsedTime(tic, toc);
	for (int cl = 0; cl < c; ++cl) {
		out1_c_l_r(cl) = cudaOut[cl];
	}
	rst::subtract(&out1_c_l_r[0], &out2_c_l_r[0], out2_c_l_r.size());
	if (rst::vectorNorm(&out1_c_l_r[0], out1_c_l_r.size(), Intrepid::NORM_ONE) > zero) {
		std::cout << "\n\nINCORRECT contractFieldFieldTensor (1): check cuda unrolled; "
		<< " diff-1norm = " << rst::vectorNorm(&out1_c_l_r[0], out1_c_l_r.size(), Intrepid::NORM_ONE) << "\n\n";
	}
	std::cout << "cuda unrolled speedup of " << elapsedTime_serial/elapsedTime_cudaUnrolled << std::endl;
	*/

#if 0
	//Warmpup
	contractFieldFieldScalarKokkos<Kokkos::OpenMP, omp_input_view_t, omp_output_view_t, omp_input_host_t, omp_output_host_t>
		(omp_hostOut, omp_hostLeft, omp_hostRight, omp_kokkosOut,
		 omp_kokkosLeft,omp_kokkosRight); clock_gettime(CLOCK_MONOTONIC, &tic);
	//repeat the calculation 5 times so we can average out some randomness
	for(int i = 0; i < 5; ++i){
		contractFieldFieldScalarKokkos<Kokkos::OpenMP, omp_input_view_t, omp_output_view_t, omp_input_host_t, omp_output_host_t>
			(omp_hostOut, omp_hostLeft, omp_hostRight, omp_kokkosOut, omp_kokkosLeft,
			 omp_kokkosRight);
	}
	clock_gettime(CLOCK_MONOTONIC, &toc);
	const double elapsedTime_kokkos = getElapsedTime(tic, toc);
	// Copy out from kokkos output view (NOT timing this)
	for (int cl = 0; cl < c; ++cl) {
		for (int lbf = 0; lbf < l; ++lbf) {
			for (int rbf = 0; rbf < r; ++rbf) {
				out1_c_l_r(cl, lbf, rbf) = omp_hostOut(cl, lbf, rbf);
			}
		}
	}
#endif

	return 0;
}